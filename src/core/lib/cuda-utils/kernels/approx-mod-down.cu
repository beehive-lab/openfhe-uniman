#include "hip/hip_runtime.h"
#include "cuda-utils/kernel-headers/approx-mod-down.cuh"

#include <cinttypes>

#include "cuda-utils/m_vectors.h"
#include "cuda-utils/kernel-headers/shared_device_functions.cuh"

__device__ inline void approxSwitchCRTBasisFunc(int ri, int ringDim, int sizeP, int sizeQ,
                                     ulong*             m_vectors, uint32_t m_vectors_sizeY,
                                     ulong*             QHatInvModq,
                                     ulong*             QHatInvModqPrecon,
                                     uint128_t*         QHatModp, uint32_t QHatModp_sizeY,
                                     uint128_t*         sum,
                                     uint128_t*         modpBarrettMu,
                                     ulong*             ans_m_vectors, uint32_t ans_sizeY) {

        if (ri == 0)
            printf("(kernel) m_vectors size = %d\n", sizeQ*ringDim);
        for(int i = 0; i < sizeQ; i++) {
            // access the data part - index ok
            //ulong xi = m_vectors[i * ringDim + ri];
            ulong xi = m_vectors[i * m_vectors_sizeY + ri];
            // access the modulus part - index ok
            //ulong qi = m_vectors[sizeQ * ringDim + i];
            ulong qi = m_vectors[sizeQ * m_vectors_sizeY + i];
            // ok
            ulong xQHatInvModqi = ModMulFastConst(xi, QHatInvModq[i], qi, QHatInvModqPrecon[i]);
            for(int j = 0; j < sizeP; j++) {
                if(ri == 0) {
                    uint128_t value = (uint128_t)QHatModp[i * QHatModp_sizeY + j];
                    uint64_t lo = (uint64_t) value;
                    uint64_t hi = (uint64_t) (value >> 64);
                    uint128_t value2 = (uint128_t)sum[ri * sizeP + j];
                    uint64_t lo2 = (uint64_t) value2;
                    uint64_t hi2 = (uint64_t) (value2 >> 64);
                    printf("cuda_ before xQHatInvModqi=%ld, QHatModp[%d]=0x%016llx%016llx, sum[%d]=0x%016llx%016llx \n", xQHatInvModqi, i * sizeP + j, (unsigned long long)hi, (unsigned long long)lo, ri * sizeP + j, (unsigned long long)hi2, (unsigned long long)lo2);
                }
                sum[ri * sizeP + j] += (uint128_t)xQHatInvModqi * QHatModp[i * QHatModp_sizeY + j];
                if(ri == 0) {
                    uint128_t value = (uint128_t)sum[ri * sizeP + j];
                    uint64_t lo2 = (uint64_t) value;
                    uint64_t hi2 = (uint64_t) (value >> 64);
                    printf("cuda_ after  sum[%d]=0x%016llx%016llx \n", ri * sizeP + j, (unsigned long long)hi2, (unsigned long long)lo2);
                }
            }
        }
        if (ri==0) {
            for (uint32_t p = 0; p < sizeP; p++) {
                uint128_t value = (uint128_t)sum[p];
                uint64_t lo = (uint64_t) value;
                uint64_t hi = (uint64_t) (value >> 64);
                printf("gpu_sum[%d] = 0x%016llx%016llx\n", p, (unsigned long long)hi, (unsigned long long)lo);
            }
        }
        for(int j = 0; j < sizeP; j++) {
            if (ri == 0)
                printf("(kernel) modulus ans_m_vectors[%d] = %llu\n", sizeP * ringDim + j, ans_m_vectors[sizeP * ringDim + j]);
            // get the modulus
            //ulong pj = ans_m_vectors[sizeP * ringDim + j];
            ulong pj = ans_m_vectors[sizeP * ans_sizeY + j];
            ans_m_vectors[j * ringDim + ri] = BarrettUint128ModUint64(sum[ri * sizeP + j], pj, modpBarrettMu[j]);
        }
}

__global__ void approxModDown(
    //scalar values
    int ringDim, int sizeQP, int sizeP, int sizeQ,
    // work data along with their column size
    ulong*      partP_m_vectors,            uint32_t partP_m_vectors_sizeY,
    uint128_t*  sum,
    ulong*      partPSwitchedToQ_m_vectors, uint32_t partPSwitchedToQ_sizeY,
    // params data along with their column size (where applicable)
    ulong*      QHatInvModq,
    ulong*      QHatInvModqPrecon,
    uint128_t*  QHatModp,                   uint32_t QHatModp_sizeY,
    uint128_t*  modpBarrettMu) {

    int ri = blockIdx.x * blockDim.x + threadIdx.x;
    if (ri < ringDim) {
        if (ri ==0) {
            printf("[kernel] partP_m_vectors[0][0]=%llu\n", partP_m_vectors[0 * ringDim + 0]);
            printf("[kernel] partP_m_vectors[0][1]=%llu\n", partP_m_vectors[0 * ringDim + 1]);
            printf("[kernel] partP_m_vectors[1][0]=%llu\n", partP_m_vectors[1 * ringDim + 0]);
            printf("[kernel] partP_m_vectors[%d][%d]=%llu\n", sizeP-1, ringDim-1, partP_m_vectors[sizeP-1 * ringDim + ringDim-1]);
        }
        // swap sizeP with sizeQ
        approxSwitchCRTBasisFunc(ri, ringDim, sizeQ, sizeP, partP_m_vectors, partP_m_vectors_sizeY, QHatInvModq, QHatInvModqPrecon, QHatModp, QHatModp_sizeY, sum, modpBarrettMu, partPSwitchedToQ_m_vectors, partPSwitchedToQ_sizeY);
    }
}

void approxModDownKernelWrapper(dim3 blocks, dim3 threads, void** args, hipStream_t stream) {
    //std::cout << "New approxSwitchCRTBasisKernelWrapper" << std::endl;
    hipError_t         cudaStatus;

    //hipDeviceSynchronize();
    cudaStatus = hipLaunchKernel((void*)approxModDown, blocks, threads, args, 0U, stream);
    if (cudaStatus != hipSuccess) {
        printf("approxModDown kernel launch failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        //return;
        exit(-1);
    }
    //hipDeviceSynchronize();

    //std::cout << "End New approxSwitchCRTBasisKernelWrapper" << std::endl;
}