#include "hip/hip_runtime.h"
/**
 * CUDA kernel for ApproxSwitchCRTBasis() function
 */

#include "cuda-utils/kernel-headers/approx-switch-crt-basis.cuh"

__device__ void initSumArray(uint128_t* sum, int sizeP) {
    for(int i=0; i<sizeP; i++) {
        sum[i] = 0;
    }
}

/**
 * CUDA implementation of:
 * NativeIntegerT ModMulFastConst(const NativeIntegerT& b, const NativeIntegerT& modulus, const NativeIntegerT& bInv)
 * from core/include/math/hal/intnat/ubintnat.h
 *
 * Validated
 */
__device__ ulong ModMulFastConst(ulong a, ulong b, ulong modulus, ulong bInv) {
    //NativeInt q      = MultDHi(this->m_value, bInv.m_value);
    ulong q = __umul64hi(a, bInv);
    //NativeInt yprime = this->m_value * b.m_value - q * modulus.m_value;
    ulong yprime = a * b - q * modulus;
    //return SignedNativeInt(yprime) - SignedNativeInt(modulus.m_value) >= 0 ? yprime - modulus.m_value : yprime;
    return (long)yprime - (long)modulus >=0 ? yprime - modulus : yprime;
}

__device__ uint128_t Mul128(ulong a, ulong b) {
    return (uint128_t)a * (uint128_t)b;
}

/**
 * add two 64-bit number with carry out, c = a + b
 * @param a: operand 1
 * @param b: operand 2
 * @param c: c = a + b
 * @return 1 if overflow occurs, 0 otherwise
 */

__device__ ulong AdditionWithCarryOut(ulong a, ulong b, ulong& c) {
    a += b;
    c = a;
    if (a < b)
        return 1;
    else
        return 0;
}

/**
 * check if adding two 64-bit number can cause overflow
 * @param a: operand 1
 * @param b: operand 2
 * @return 1 if overflow occurs, 0 otherwise
 */
__device__ ulong IsAdditionOverflow(ulong a, ulong b) {
    a += b;
    if (a < b)
        return 1;
    else
        return 0;
}

/**
 * Barrett reduction of 128-bit integer modulo 64-bit integer. Source: Menezes,
 * Alfred; Oorschot, Paul; Vanstone, Scott. Handbook of Applied Cryptography,
 * Section 14.3.3.
 * @param a: operand (128-bit)
 * @param m: modulus (64-bit)
 * @param mu: 2^128/modulus (128-bit)
 * @return result: 64-bit result = a mod m
 */
__device__ ulong BarrettUint128ModUint64(uint128_t a, ulong modulus, uint128_t mu) {
    // (a * mu)/2^128 // we need the upper 128-bit of (256-bit product)
    ulong result = 0, a_lo = 0, a_hi = 0, mu_lo = 0, mu_hi = 0, left_hi = 0, middle_lo = 0, middle_hi = 0, tmp1 = 0,
          tmp2 = 0, carry = 0;
    uint128_t middle = 0;

    a_lo  = (uint64_t)a;
    a_hi  = a >> 64;
    mu_lo = (uint64_t)mu;
    mu_hi = mu >> 64;

    left_hi = (Mul128(a_lo, mu_lo)) >> 64;  // mul left parts, discard lower word

    middle    = Mul128(a_lo, mu_hi);  // mul middle first
    middle_lo = (uint64_t)middle;
    middle_hi = middle >> 64;

    // accumulate and check carry
    carry = AdditionWithCarryOut(middle_lo, left_hi, tmp1);

    tmp2 = middle_hi + carry;  // accumulate

    middle    = Mul128(a_hi, mu_lo);  // mul middle second
    middle_lo = (uint64_t)middle;
    middle_hi = middle >> 64;

    carry = IsAdditionOverflow(middle_lo, tmp1);  // check carry

    left_hi = middle_hi + carry;  // accumulate

    // now we have the lower word of (a * mu)/2^128, no need for higher word
    tmp1 = a_hi * mu_hi + tmp2 + left_hi;

    // subtract lower words only, higher words should be the same
    result = a_lo - tmp1 * modulus;

    while (result >= modulus)
        result -= modulus;

    return result;
}

__global__ void approxSwitchCRTBasis(int ringDim, int sizeP, int sizeQ,
                                     ulong*             m_vectors_data,
                                     ulong*             m_vectors_modulus,
                                     ulong*             QHatInvModq,
                                     ulong*             QHatInvModqPrecon,
                                     uint128_t*         QHatModp,
                                     uint128_t*         sum,
                                     uint128_t*         modpBarrettMu,
                                     ulong*             ans_m_vectors_data,
                                     ulong*             ans_m_vectors_modulus) {

    int ri = blockIdx.x * blockDim.x + threadIdx.x;
    //for(int ri = 0; ri < ringDim; ri++) {
    if (ri < ringDim) {
        //__int128 sum[sizeP];
        //initSumArray(sum, sizeP);
        for(int i = 0; i < sizeQ; i++) {
            //const NativeInteger& xi     = m_vectors[i][ri];
            ulong xi = m_vectors_data[(i * ringDim) + ri];
            //const NativeInteger& qi     = m_vectors[i].GetModulus();
            ulong qi = m_vectors_modulus[i];
            //NativeInteger xQHatInvModqi = xi.ModMulFastConst(QHatInvModq[i], qi, QHatInvModqPrecon[i]);
            ulong xQHatInvModqi = ModMulFastConst(xi, QHatInvModq[i], qi, QHatInvModqPrecon[i]);
            // debugging: check ModMulFastConst - ok
            /*if(ri < 2) {
                printf("cuda_ [%d][%d]: ModMulFastConst(%ld, %ld, %ld) = %ld\n", i, ri, xi, QHatInvModq[i],qi, xQHatInvModqi);
            }*/
            for(int j = 0; j < sizeP; j++) {
                // sum[j] += Mul128(xQHatInvModqi.ConvertToInt(), QHatModp[i][j].ConvertToInt());
                sum[ri * sizeP + j] += (uint128_t)xQHatInvModqi * QHatModp[i * sizeP + j];
            }
        }

        for(int j = 0; j < sizeP; j++) {
            //const NativeInteger& pj = ans.m_vectors[j].GetModulus();
            ulong pj = ans_m_vectors_modulus[j];
            //ans.m_vectors[j][ri]    = BarrettUint128ModUint64(sum[j], pj.ConvertToInt(), modpBarrettMu[j]);
            ans_m_vectors_data[(j * ringDim) + ri] = BarrettUint128ModUint64(sum[ri * sizeP + j], pj, modpBarrettMu[j]);
            //ans_m_vectors[j].data[ri] = ri;
        }
    }
}

void approxSwitchCRTBasisKernelWrapper(dim3 blocks, dim3 threads, void** args, hipStream_t stream) {
    //std::cout << "New approxSwitchCRTBasisKernelWrapper" << std::endl;
    hipError_t         cudaStatus;

    // Calculate resources needed
    int smCount;
    hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, 0);

    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(approxSwitchCRTBasis));

    // Calculate the total number of registers and shared memory usage
    int totalThreadsPerBlock = threads.x;
    int sharedMemPerBlock = attr.sharedSizeBytes;
    int numRegsPerThread = attr.numRegs;

    //std::cout << "Total threads per block: " << totalThreadsPerBlock << std::endl;
    //std::cout << "Total shared memory per block: " << sharedMemPerBlock << " bytes" << std::endl;
    //std::cout << "Number of registers per thread: " << numRegsPerThread << std::endl;
    //std::cout << "Max threads per multiprocessor: " << attr.maxThreadsPerBlock << std::endl;
    //std::cout << "Number of multiprocessors: " << smCount << std::endl;

    //hipDeviceSynchronize();
    cudaStatus = hipLaunchKernel((void*)approxSwitchCRTBasis, blocks, threads, args, 0U, stream);
    if (cudaStatus != hipSuccess) {
        printf("approxSwitchCRTBasis kernel launch failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        //return;
        exit(-1);
    }
    //hipDeviceSynchronize();

    //std::cout << "End New approxSwitchCRTBasisKernelWrapper" << std::endl;

}

void callApproxSwitchCRTBasisKernel(int gpuBlocks, int gpuThreads,
                                    int ringDim, int sizeP, int sizeQ,
                                    m_vectors_struct*   host_m_vectors,
                                    ulong*              host_QHatInvModq,
                                    ulong*              host_QHatInvModqPrecon,
                                    uint128_t*          host_QHatModp,
                                    uint128_t*          host_modpBarrettMu,
                                    m_vectors_struct*   host_ans_m_vectors) {

    std::cout << "[callApproxSwitchCRTBasisKernel]: sizeP = " << sizeP << ", sizeQ = " << sizeQ << std::endl;

    // debugging:
    //std::cout << "==> callApproxSwitchCRTBasisKernel" << std::endl;

    hipError_t         cudaStatus;

    m_vectors_struct*   device_m_vectors;
    ulong*              device_QHatInvModq;
    ulong*              device_QHatInvModqPrecon;
    uint128_t*          device_QHatModp;
    uint128_t*          device_sum;
    uint128_t*          device_modpBarrettMu;
    m_vectors_struct*   device_ans_m_vectors;

    // m_vectors
    // inspired by: https://stackoverflow.com/questions/30082991/memory-allocation-on-gpu-for-dynamic-array-of-structs
    hipMalloc((void**)&device_m_vectors, sizeQ * sizeof(m_vectors_struct));
    hipMemcpy(device_m_vectors, host_m_vectors, sizeQ * sizeof(m_vectors_struct), hipMemcpyHostToDevice);

    unsigned long* tmp_data[sizeQ];

    for (int q = 0; q < sizeQ; ++q) {
        hipMalloc((void**)&(tmp_data[q]), ringDim * sizeof(unsigned long));
        hipMemcpy(&(device_m_vectors[q].data), &(tmp_data[q]), sizeof(unsigned long*), hipMemcpyHostToDevice);
        hipMemcpy(tmp_data[q], host_m_vectors[q].data, ringDim * sizeof(unsigned long), hipMemcpyHostToDevice);
    }

    // qhatinvmodq
    hipMalloc((void**)&device_QHatInvModq, sizeQ * sizeof(unsigned long));
    hipMemcpy(device_QHatInvModq, host_QHatInvModq, sizeQ * sizeof(unsigned long), hipMemcpyHostToDevice);

    // QHatInvModqPrecon
    hipMalloc((void**)&device_QHatInvModqPrecon, sizeQ * sizeof(unsigned long));
    hipMemcpy(device_QHatInvModqPrecon, host_QHatInvModqPrecon, sizeQ * sizeof(unsigned long), hipMemcpyHostToDevice);

    // qhatmodp
    hipMalloc((void**)&device_QHatModp,    sizeQ * sizeP * sizeof(uint128_t));
    hipMemcpy(device_QHatModp, host_QHatModp, sizeQ * sizeP * sizeof(uint128_t), hipMemcpyHostToDevice);

    // sum
    hipMalloc((void**)&device_sum,         sizeP * ringDim * sizeof(uint128_t));
    hipMemset(device_sum, 0, sizeP * ringDim * sizeof(uint128_t));

    // modpBarrettMu
    hipMalloc((void**)&device_modpBarrettMu, sizeP * sizeof(uint128_t));
    hipMemcpy(device_modpBarrettMu, host_modpBarrettMu, sizeP * sizeof(uint128_t), hipMemcpyHostToDevice);

    // ans_m_vectors
    hipMalloc((void**)&device_ans_m_vectors, sizeP * sizeof(m_vectors_struct));
    hipMemcpy(device_ans_m_vectors, host_ans_m_vectors, sizeP * sizeof(m_vectors_struct), hipMemcpyHostToDevice);

    unsigned long* tmp_device_ans_m_vectors_data[sizeP];

    for (int p = 0; p < sizeP; ++p) {
        hipMalloc((void**)&(tmp_device_ans_m_vectors_data[p]), ringDim * sizeof(unsigned long));
        hipMemcpy(&(device_ans_m_vectors[p].data), &(tmp_device_ans_m_vectors_data[p]), sizeof(unsigned long*), hipMemcpyHostToDevice);
        //hipMemcpy(tmp_data[q], host_m_vectors[q].data, ringDim * sizeof(unsigned long), hipMemcpyHostToDevice);
    }


    // hipLaunchKernel
    //dim3 blocks = dim3(1U, 1U, 1U); // Set the grid dimensions
    //hipOccupancyMaxActiveBlocksPerMultiprocessor
    dim3 blocks = dim3(gpuBlocks, 1U, 1U); // Set the grid dimensions
    dim3 threads = dim3(gpuThreads, 1U, 1U); // Set the block dimensions
    void *args[] = {&ringDim, &sizeP, &sizeQ, &device_m_vectors, &device_QHatInvModq, &device_QHatInvModqPrecon, &device_QHatModp, &device_sum, &device_modpBarrettMu, &device_ans_m_vectors};
    // debugging:
    // printf("Before kernel launch\n");
    cudaStatus = hipLaunchKernel((void*)approxSwitchCRTBasis, blocks, threads, args, 0U, nullptr);
    if (cudaStatus != hipSuccess) {
        printf("approxSwitchCRTBasis kernel launch failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        return;
    }
    hipDeviceSynchronize();
    // debugging:
    //printf("After kernel launch\n");

    // copy out the result ans vector
    for(int p = 0; p < sizeP; p++) {
        hipMemcpy(host_ans_m_vectors[p].data, tmp_device_ans_m_vectors_data[p], ringDim * sizeof(unsigned long), hipMemcpyDeviceToHost);
    }

    // debugging: print sum result
    /*printf("gpu_sum size = %d\n", sizeP);
    for(int i = 0; i < sizeP; i++) {
        //printf("host_sum[%d] = %llx\n", i, host_sum[i]);
        uint64_t lo = (uint64_t) host_sum[i];
        uint64_t hi = (uint64_t) (host_sum[i] >> 64);
        printf("gpu_sum[%d] = 0x%016llx%016llx\n", i, (unsigned long long)hi, (unsigned long long)lo);
    }*/

    // debugging: print ans_m_vectors result -ok
    /*int tmp_ri = ringDim-1;
    for(int p = 0; p < sizeP; p++) {
        std::cout << "gpu_ans_m_vectors[" << p << ", " << tmp_ri << "] = " << host_ans_m_vectors[p].data[tmp_ri] << std::endl;
    }*/

    hipFree(device_m_vectors);
    hipFree(device_QHatInvModq);
    hipFree(device_QHatInvModqPrecon);
    hipFree(device_QHatModp);
    hipFree(device_sum);
    hipFree(device_modpBarrettMu);
    hipFree(device_ans_m_vectors);

    //std::cout << "END Old callApproxSwitchCRTBasisKernel" << std::endl;

}

void printMemoryInfo() {
    size_t freeMem;
    size_t totalMem;

    hipError_t err = hipMemGetInfo(&freeMem, &totalMem);
    if (err != hipSuccess) {
        std::cerr << "Error getting memory info: " << hipGetErrorString(err) << std::endl;
        return;
    }

    std::cout << "Total device memory: " << totalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Free device memory: " << freeMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Used device memory: " << (totalMem - freeMem) / (1024 * 1024) << " MB" << std::endl;
}


/**
 * A dummy CUDA kernel.
 */
static __global__ void myKernel(int* data, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        data[tid] += 1;
    }
}

/**
 * Call the dummy kernel.
 */
void callMyKernel(uint32_t ringDim, uint32_t sizeQ, uint32_t sizeP) {

    // here
    int n = 32;
    int i;
    int* h_data;   // host data
    int* d_data;   // device data
    hipError_t cudaStatus;

    h_data = (int*)malloc(n * sizeof(int));
    for(i=0; i < n; i++) {
        h_data[i] = 1;
    }
    ///////////////

    ///////////////

    cudaStatus = hipMalloc((void **)&d_data, n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        return;
    }

    cudaStatus = hipMemcpy(d_data, h_data, n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        return;
    }

    dim3 blocks = dim3(1U, 1U, 1U); // Set the grid dimensions
    dim3 threads = dim3(n, 1U, 1U); // Set the block dimensions
    void *args[] = { &d_data, &n};
    printf("Before kernel launch\n");
    cudaStatus = hipLaunchKernel((void*)myKernel, blocks, threads, args, 0U, nullptr);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        return;
    }
    hipDeviceSynchronize();
    printf("After kernel launch\n");
    hipMemcpy(h_data, d_data, n * sizeof(int), hipMemcpyDeviceToHost);

    // print result
    for(i = 0; i < n; i++) {
        printf("h_data[%d] = %d\n", i, h_data[i]);
    }

    hipFree(d_data);
    free(h_data);
}