/**
 * CUDA kernel for ApproxSwitchCRTBasis() function
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint> // for uint32_t type

static __global__ void myKernel(int* data, int n) {
    __int128 a = 0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        data[tid] += 1;
    }
}

void callMyKernel(uint32_t ringDim, uint32_t sizeQ, uint32_t sizeP) {

    // here
    int n = 32;
    int i;
    int* h_data;   // host data
    int* d_data;   // device data
    hipError_t cudaStatus;

    h_data = (int*)malloc(n * sizeof(int));
    for(i=0; i < n; i++) {
        h_data[i] = 1;
    }
    ///////////////

    ///////////////

    cudaStatus = hipMalloc((void **)&d_data, n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        return;
    }

    cudaStatus = hipMemcpy(d_data, h_data, n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        return;
    }

    dim3 blocks = dim3(1U, 1U, 1U); // Set the grid dimensions
    dim3 threads = dim3(n, 1U, 1U); // Set the block dimensions
    void *args[] = { &d_data, &n};
    printf("Before kernel launch\n");
    cudaStatus = hipLaunchKernel(reinterpret_cast<const void*>((void*)myKernel), blocks, threads, args, 0U, nullptr);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s (%d) \n", hipGetErrorString(cudaStatus), cudaStatus);
        return;
    }
    hipDeviceSynchronize();
    printf("After kernel launch\n");
    hipMemcpy(h_data, d_data, n * sizeof(int), hipMemcpyDeviceToHost);

    // print result
    for(i = 0; i < n; i++) {
        printf("h_data[%d] = %d\n", i, h_data[i]);
    }

    hipFree(d_data);
    free(h_data);
}